// compile for python
// nvcc -shared -o stream_manage.so stream_manage.cu -L/usr/local/cuda/lib64 -lcudart

// compile for C++
// nvcc add_floats.c stream_manage.cu -o stream_manage -L/usr/local/cuda/lib64 -lcudart
// ./stream_manage

#include <stdio.h>
#include "add_floats.h"

extern "C" hipStream_t createStream() {
    hipStream_t stream;
    hipStreamCreate(&stream);
    return stream;
}

extern "C" void copyCpuToGpuOnStream(int8_t *dst, const int8_t *src, int n, hipStream_t stream) {
    hipMemcpyAsync(dst, src, n * sizeof(int8_t), hipMemcpyHostToDevice, stream);
}

extern "C" void copyGpuToCpuOnStream(int8_t *dst, const int8_t *src, int n, hipStream_t stream) {
    hipMemcpyAsync(dst, src, n * sizeof(int8_t), hipMemcpyDeviceToHost, stream);
}

extern "C" void copy2DTensorCpuToGpuOnStream(int8_t *dst, const int8_t *src, int rows, int cols, hipStream_t stream) {
    size_t size = rows * cols * sizeof(int8_t);
    hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

extern "C" void copy2DTensorGpuToCpuOnStream(int8_t *dst, const int8_t *src, int rows, int cols, hipStream_t stream) {
    size_t size = rows * cols * sizeof(int8_t);
    hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream);
}

extern "C" void copyCpuToGpuOnStream_float(float *dst, const float *src, int n, hipStream_t stream) {
    hipMemcpyAsync(dst, src, n * sizeof(float), hipMemcpyHostToDevice, stream);
}

extern "C" void copyGpuToCpuOnStream_float(float *dst, const float *src, int n, hipStream_t stream) {
    hipMemcpyAsync(dst, src, n * sizeof(float), hipMemcpyDeviceToHost, stream);
}

extern "C" void copy2DTensorCpuToGpuOnStream_float(float *dst, const float *src, int rows, int cols, hipStream_t stream) {
    size_t size = rows * cols * sizeof(float);
    hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

extern "C" void copy2DTensorGpuToCpuOnStream_float(float *dst, const float *src, int rows, int cols, hipStream_t stream) {
    size_t size = rows * cols * sizeof(float);
    hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream);
}

extern "C" void synchronizeStream(hipStream_t stream) {
    hipStreamSynchronize(stream);
}

extern "C" void destroyStream(hipStream_t stream) {
    hipStreamDestroy(stream);
}

void float_operate(){
    float a[] = {1.1, 2.2, 3.3};
    float b[] = {4.4, 5.5, 6.6};
    float result[3];

    float *d_result; // Device pointer for the result array

    // Calculate the sum of the float arrays on the CPU
    addFloats(a, b, result, 3);

    // Allocate memory on the device
    hipMalloc((void **)&d_result, sizeof(result));

    hipStream_t stream;
    hipStreamCreate(&stream);
    copyCpuToGpuOnStream_float(d_result, result, 3, stream);

    // Output the results on the host
    printf("Result of adding two float arrays:\n");
    for (int i = 0; i < 3; i++) {
        printf("%f + %f = %f\n", a[i], b[i], result[i]);
    }

    copyGpuToCpuOnStream_float(result, d_result, 3, stream);
    // Cleanup
    hipFree(d_result);
    hipStreamDestroy(stream);
}

int main(){
    float_operate();
}
